#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <cstdint>

////////////////////////////////////////////////////////////////////////////////
// Simple Memcpy Kernel
////////////////////////////////////////////////////////////////////////////////
template <int TILE> __global__ void memcpy(float *dst, const float *src) {
    int iblock = blockIdx.x + blockIdx.y * gridDim.x;
    int index = threadIdx.x + TILE * iblock * blockDim.x;

    float a[TILE];

#pragma unroll
    for (int i = 0; i < TILE; i++) {
        a[i] = src[index + i * blockDim.x];
    }

#pragma unroll
    for (int i = 0; i < TILE; i++) {
        dst[index + i * blockDim.x] = a[i];
    }

//     int iblock = blockIdx.x + blockIdx.y * gridDim.x;
//     int index = threadIdx.x + TILE * iblock * blockDim.x;

//     float4* dst4 = reinterpret_cast<float4*>(dst);
//     const float4* src4 = reinterpret_cast<const float4*>(src);
    
//     const int TILE4 = TILE / 4;
//     float4 a[TILE4];

//     printf("Kernel configuration: TILE=%d\n", TILE);

// #pragma unroll
//     for (int i = 0; i < TILE4; i++) {
//         a[i] = src4[index + i * blockDim.x];
//     }

// #pragma unroll
//     for (int i = 0; i < TILE4; i++) {
//         dst4[index + i * blockDim.x] = a[i];
//     }
}


////////////////////////////////////////////////////////////////////////////////
// Prelab Question 3: Fill in the shared memory sizes you want to run the
// kernel with. Changing these values will limit the occupancy of the kernel.
////////////////////////////////////////////////////////////////////////////////
inline std::vector<int> shared_memory_configuration() { 
    return {19000}; 
    // return {7000}; 
    // return {5000}; 
    // return {0}; 
}

////////////////////////////////////////////////////////////////////////////////
// Main
////////////////////////////////////////////////////////////////////////////////
int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int size = 100 * 1024 * 1024;
    float *src, *dst;
    hipMalloc(&src, size * sizeof(float));
    hipMalloc(&dst, size * sizeof(float));
    
    // Verify 16-byte alignment for float4 operations
    if ((uintptr_t)src % 16 != 0 || (uintptr_t)dst % 16 != 0) {
        printf("Warning: Memory not 16-byte aligned! src=%p, dst=%p\n", src, dst);
    }

    // Host buffer for L2 invalidation
    float *h_src = (float *)malloc(size * sizeof(float));

    const int TILE = 8; // TILE size for memcpy kernel
    const int threads = 64;
    const int blocks = (size + TILE * threads - 1) / (TILE * threads);

    // Shared memory configurations
    std::vector<int> smem_sizes = shared_memory_configuration();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Print table header
    printf(
        "\n%-12s %-10s %-12s %-12s %-10s %-10s %-14s\n",
        "SharedMem",
        "Time(ms)",
        "BW(GB/s)",
        "Eff(%)",
        "Occ(%)",
        "Blocks/SM",
        "Bytes in flight");
    printf("-----------------------------------------------------------------------------"
           "----------------\n");

    hipFuncSetAttribute(reinterpret_cast<const void*>(
        memcpy<TILE>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        prop.sharedMemPerBlockOptin);

    for (int shared_mem : smem_sizes) {

        // Benchmark
        hipEventRecord(start);
        memcpy<TILE><<<blocks, threads, shared_mem>>>(dst, src);
        // test return value
        // printf("return value: %d\n", hipGetLastError());
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float time;
        hipEventElapsedTime(&time, start, stop);

        double bytes = 2.0 * size * sizeof(float); // read + write
        double bw = bytes * 1000.0 / (time * 1e9);

        int numBlocks;
        hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &numBlocks,
            memcpy<TILE>,
            threads,
            shared_mem);

        float occupancy = ((numBlocks * threads / 32) / 48.0f) * 100.0f;
        int bytes_per_thread = TILE * (int)sizeof(float) * 48 * 64 * numBlocks;

        printf(
            "%-12d %-10.3f %-12.3f %-12.1f %-10.1f %-10d %-14d\n",
            shared_mem,
            time,
            bw,
            100.0 * bw / (2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1e6),
            occupancy,
            numBlocks,
            bytes_per_thread);
        // Invalidate L2 cache by copying "fresh" data to device
        for (int i = 0; i < size; i++) {
            h_src[i] = (float)(i % 997) * 0.123f;
        }
        hipMemcpy(src, h_src, size * sizeof(float), hipMemcpyHostToDevice);
    }

    free(h_src);
    hipFree(src);
    hipFree(dst);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
